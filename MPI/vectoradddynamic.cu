#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define N 1024 * 1024
#define THREADS_PER_BLOCK 256

__global__ void vectorAddKernel(float* A, float* B, float* C) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    size_t size = N * sizeof(float);
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    vectorAddKernel<<<blocks, THREADS_PER_BLOCK>>>(d_A, d_B, d_C);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    float memClockKHz = static_cast<float>(prop.memoryClockRate);
    float memBusWidth = static_cast<float>(prop.memoryBusWidth);

    float theoreticalBW = 2.0f * memClockKHz * 1000 * (memBusWidth / 8.0f) / (1 << 30);
    std::cout << "Theoretical Memory Bandwidth: " << theoreticalBW << " GB/s" << std::endl;

    float RBytes = size * 2;
    float WBytes = size;
    float timeInSeconds = milliseconds / 1000.0f;

    float measuredBW = (RBytes + WBytes) / (timeInSeconds * (1 << 30));
    std::cout << "Measured Memory Bandwidth: " << measuredBW << " GB/s" << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
